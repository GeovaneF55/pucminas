/*
    Aluno: Geovane Fonseca de Sousa Santos
    Matrícula: 553237
    Matéria: Computação Paralela
    Tarefa 26: Padrão SCAN em CUDA

    SEQUENCIAL:

        real	0m0.425s
        user	0m0.195s
        sys	0m0.226s

    PARALELO CUDA:

        real	0m2.072s
        user	0m0.849s
        sys	0m1.126s
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void scan_cuda(double* a, double *s, int width) {
  // kernel scan
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
  double x;

  // cria vetor na memória local
  __shared__ double p[1024];
  
  // carrega elementos do vetor da memória global para a local
  if(b+t < width) {
    p[t] = a[b+t];
  }

  // espera que todas as threads tenham carregado seus elementos
  __syncthreads();

  // realiza o scan em log n passos
  for (int i = 1; i < blockDim.x; i *= 2) {

    // verifica se a thread ainda participa neste passo
    if(t >= i){
      x = p[t] + p[t-i]; // atribui a soma para uma variável temporária
    }

    __syncthreads(); // espera threads fazerem as somas

    // se for a última thread do bloco
    if(t >= i) {
      p[t] = x; // copia a soma em definitivo para o vetor local
    }

    __syncthreads();
  }

  // copia da memória local para a global
  if(b + t < width) {
    a[b+t] = p[t];
  }

  // se for a última thread do bloco
  if(t == blockDim.x-1) {
    s[blockIdx.x+1] = a[b+t]; // copia o seu valor para o vetor de saída
  }
} 

__global__ void add_cuda(double *a, double *s, int width) {
  // kernel soma
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;

  // soma o somatório do último elemento do bloco anterior ao elemento atual
  if(b+t < width)
    a[b+t] += s[blockIdx.x];
}

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocar vetores "a" e "s" no device
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_s, s_size);

  // copiar vetor "a" para o device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  // definição do número de blocos e threads (dimGrid e dimBlock)
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);

  // chamada do kernel scan
  scan_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // copiar vetor "s" para o host
  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);

  // scan no host (já implementado)
  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];
 
  // copiar vetor "s" para o device
  hipMemcpy(d_s, s, s_size, hipMemcpyHostToDevice);

  // chamada do kernel da soma
  add_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // copiar o vetor "a" para o host
  hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

  printf("\na[%d] = %f\n",width-1,a[width-1]);
  
  hipFree(d_a);
  hipFree(d_s);
}
