
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void scan_cuda(double* a, double *s, int width) {
  // kernel scan
} 

__global__ void add_cuda(double *a, double *s, int width) {
  // kernel soma
}

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocar vetores "a" e "s" no device

  // copiar vetor "a" para o device

  // definição do número de blocos e threads (dimGrid e dimBlock)

  // chamada do kernel scan

  // copiar vetor "s" para o host

  // scan no host (já implementado)
  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];
 
  // copiar vetor "s" para o device

  // chamada do kernel da soma

  // copiar o vetor "a" para o host

  printf("\na[%d] = %f\n",width-1,a[width-1]);
  
  hipFree(d_a);
  hipFree(d_s);
}
