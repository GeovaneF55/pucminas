/*
	Aluno: Geovane Fonseca de Sousa Santos
	Matrícula: 553237
	Matéria: Coputação Paralela
	Tarefa 24: Somatório em CUDA

	Sequencial:

	real	0m0.300s
	user	0m0.068s
	sys	0m0.229s

	Paralelo para multicore em OpenMP:

	real	0m0.148s
	user	0m0.094s
	sys	0m0.408s

	Paralelo para GPU com OpenMP:

	real	0m1.130s
	user	0m0.099s
	sys	0m1.285s

	CUDA:

	real	0m1.608s
	user	0m0.535s
	sys	0m0.960s

	==5588== Profiling application: ./sum_cuda
	==5588== Profiling result:
	Time(%)      Time     Calls       Avg       Min       Max  Name
	 95.54%  468.59ms         1  468.59ms  468.59ms  468.59ms  [CUDA memcpy HtoD]
	  4.39%  21.529ms         1  21.529ms  21.529ms  21.529ms  sum_cuda(double*, double*, int)
	  0.07%  362.22us         1  362.22us  362.22us  362.22us  [CUDA memcpy DtoH]

	==5588== API calls:
	Time(%)      Time     Calls       Avg       Min       Max  Name
	 62.79%  492.67ms         2  246.33ms  25.263ms  467.41ms  cudaMemcpy
	 36.71%  288.05ms         2  144.02ms  32.580us  288.01ms  cudaMalloc
	  0.42%  3.2584ms         2  1.6292ms  29.610us  3.2288ms  cudaFree
	  0.06%  484.80us        90  5.3860us     292ns  206.45us  cuDeviceGetAttribute
	  0.01%  88.564us         1  88.564us  88.564us  88.564us  cuDeviceTotalMem
	  0.01%  67.905us         1  67.905us  67.905us  67.905us  cuDeviceGetName
	  0.01%  52.107us         1  52.107us  52.107us  52.107us  cudaLaunch
	  0.00%  9.7980us         3  3.2660us     371ns  7.6310us  cudaSetupArgument
	  0.00%  2.9470us         2  1.4730us     964ns  1.9830us  cuDeviceGetCount
	  0.00%  1.7250us         1  1.7250us  1.7250us  1.7250us  cudaConfigureCall
	  0.00%  1.1440us         2     572ns     544ns     600ns  cuDeviceGet

	RESULTADOS:

	* O resultado sequencial é considerado o resultado base para fazer os cálculos

	* O resultado paralelo para multicore em OpenMP teve um speedup de (0.300/0.148) = 2,027

	* O resultado paralelo para GPU com OpenMP não obteve speedup em relação ao sequencial devido ao overhead de enviar os dados para a GPU

	* O resultado no CUDA foi pior ainda, também por causa do overhead de enviar os dados para a GPU e por alocá-los
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sum_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;

  __shared__ double o[1024];  

  if(b+t < width)
    o[t] = a[b+t];

  __syncthreads();
  
  int i;
  for(i = blockDim.x/2; i > 0; i /= 2) {
    if(t < i && b+t+i < width)
      o[t] += o[t+i];
    
  	__syncthreads();
  }

  if(t == 0)
    s[blockIdx.x] = o[0];  
} 

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocação e cópia dos dados
  hipMalloc((void **) &d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_s, s_size);

  // definição do número de blocos e threads
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);

  // chamada do kernel
  sum_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // cópia dos resultados para o host
  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);

  // soma das reduções parciais
  for(int i = 1; i < num_blocks; i++) 
    s[0] += s[i];

  printf("\nSum = %f\n",s[0]);
  
  hipFree(d_a);
  hipFree(d_s);
}
